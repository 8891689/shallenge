#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2024 Nol Moonen

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to furnish the Software to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.

Copyright (c) 2025 8891689

This code file contains modifications of the original work (Copyright (c) 2024 Nol Moonen).
Modifications include but are not limited to: manual expansion optimization of SHA-256 core functions, using CUDA constant memory to store constants and fixed messages, separation of SHA-256 implementations between host and device, and improvements in block-wise reduction logic.
*/

#include <cub/block/block_reduce.cuh>
#include <hip/hip_runtime.h>

#include <cassert>
#include <cstdlib>
#include <cstring>
#include <limits>
#include <stdint.h>
#include <stdio.h>
#include <vector>
#include <cstdio> // 中文标注: 添加了cstdio头文件，用于fprintf

// 中文标注: 移除了原始码中的 DEVICE_UNROLL 宏定义，因为核心 SHA-256 循环将手动展开

// SHA-256 Constants (part of the algorithm)
// Declare device-side constants for arrays
// 中文标注: 将常数 K 分离为设备端 d_K (__constant__) 和主机端 h_K (constexpr)，并在main函数中拷贝
__constant__ uint32_t d_K[64];

// Initial hash values (part of the algorithm) - Use just constexpr for scalars
// 中文标注: 将初始哈希值 aa-hh 重命名为 H0-H7，仍为 constexpr
constexpr uint32_t H0 = 0x6a09e667;
constexpr uint32_t H1 = 0xbb67ae85;
constexpr uint32_t H2 = 0x3c6ef372;
constexpr uint32_t H3 = 0xa54ff53a;
constexpr uint32_t H4 = 0x510e527f;
constexpr uint32_t H5 = 0x9b05688c;
constexpr uint32_t H6 = 0x1f83d9ab;
constexpr uint32_t H7 = 0x5be0cd19;

// Host-side versions of K for copying (constexpr array)
// 中文标注: 这是主机端使用的 K 常数数组，用于拷贝到设备端
constexpr uint32_t h_K[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4,
    0xab1c5ed5, 0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe,
    0x9bdc06a7, 0xc19bf174, 0xe49b69c1, /* REMOVED DUPLICATE in original */ 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f,
    0x4a7484aa, 0x5cb0a9dc, 0x76f988da, 0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7,
    0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967, 0x27b70a85, 0x2e1b2138, 0x4d2c6dfc,
    0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85, 0xa2bfe8a1, 0xa81a664b,
    0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070, 0x19a4c116,
    0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7,
    0xc67178f2};


struct nonce_t {
    uint32_t m11;
    uint32_t m12;
    uint32_t m13;
};

struct hash_t {
    uint32_t arr[8];
};

// Basic SHA-256 bitwise functions as inline functions (for both host and device)
// 中文标注: 将位操作宏替换为 __forceinline__ __host__ __device__ 函数
__forceinline__ __host__ __device__ uint32_t rotr(uint32_t x, uint32_t n) {
    return (x >> n) | (x << (32 - n));
}

__forceinline__ __host__ __device__ uint32_t maj(uint32_t x, uint32_t y, uint32_t z) {
    return (x & y) | (z & (x ^ y));
}

__forceinline__ __host__ __device__ uint32_t ch(uint32_t x, uint32_t y, uint32_t z) {
    return (z ^ (x & (y ^ z)));
}

__forceinline__ __host__ __device__ uint32_t ep0(uint32_t x) {
    return rotr(x, 2) ^ rotr(x, 13) ^ rotr(x, 22);
}

__forceinline__ __host__ __device__ uint32_t ep1(uint32_t x) {
    return rotr(x, 6) ^ rotr(x, 11) ^ rotr(x, 25);
}

__forceinline__ __host__ __device__ uint32_t sig0(uint32_t x) {
    return rotr(x, 7) ^ rotr(x, 18) ^ (x >> 3);
}

__forceinline__ __host__ __device__ uint32_t sig1(uint32_t x) {
    return rotr(x, 17) ^ rotr(x, 19) ^ (x >> 10);
}

// Optimized Endian Swap (same as original, essential for performance)
__forceinline__ __device__ __host__ uint32_t swap_endian(uint32_t x)
{
#ifdef __CUDA_ARCH__
    // Use efficient device intrinsic
    return __byte_perm(x, (uint32_t)0, (uint32_t)0x0123);
#else
    // Standard host implementation
    const uint8_t* ptr = reinterpret_cast<const uint8_t*>(&x);
    return uint32_t{ptr[3]} | (uint32_t{ptr[2]} << 8) | (uint32_t{ptr[1]} << 16) |
           (uint32_t{ptr[0]} << 24);
#endif
}

// Fixed parts of the message block (same as original, defined by the problem)
// Declare device-side constants for arrays
// 中文标注: 将固定消息部分分离为设备端 d_fixed_msg (__constant__) 和主机端 h_fixed_msg (constexpr)，并在main函数中拷贝
__constant__ uint32_t d_fixed_msg[11];

// Host-side versions of fixed_msg for copying (constexpr array)
// 中文标注: 这是主机端使用的固定消息数组，用于拷贝到设备端
constexpr uint32_t h_fixed_msg[11] = {
    0x6e6f6c2f, // 'nol/'
    0x30303030, // '0000' * 10
    0x30303030,
    0x30303030,
    0x30303030,
    0x30303030,
    0x30303030,
    0x30303030,
    0x30303030,
    0x30303030,
    0x30303030
};

// Padding for SHA-256 (message length in bits) - Use just constexpr for scalars
// 中文标注: 填充常数 m14 和 m15 重命名为 padding_m14 和 padding_m15
constexpr uint32_t padding_m14 = 0x00000000; // upper part of u64 size
constexpr uint32_t padding_m15 = 0x000001b8; // length, 55 bytes = 440 bits (0x1b8)

// Helper macro for the core SHA-256 transformation step
// Using do-while(0) to create a single statement block for local variable scoping
// 中文标注: SHA-256 核心变换步骤宏
#define SHA256_TRANSFORM_STEP(a, b, c, d, e, f, g, h, w_i, k_i) do { \
    uint32_t T1 = h + ep1(e) + ch(e, f, g) + k_i + w_i;   \
    uint32_t T2 = ep0(a) + maj(a, b, c);                  \
    h = g; g = f; f = e; e = d + T1;                      \
    d = c; c = b; b = a; a = T1 + T2;                     \
} while(0)


// SHA-256 compression function implementation for DEVICE - Manual Unrolling
// 中文标注: 设备端 SHA-256 压缩函数实现 - 手动展开
__forceinline__ __device__ void sha256_manual_unroll_device(
    hash_t& hash, uint32_t m11, uint32_t m12, uint32_t m13)
{
    uint32_t a = H0, b = H1, c = H2, d = H3, e = H4, f = H5, g = H6, h = H7; // 中文标注: 使用重命名后的初始哈希值

    uint32_t W[16]; // Circular buffer for message schedule

    // Load initial 16 words into the buffer
    // 中文标注: 从设备端常量载入固定消息部分
    W[0] = d_fixed_msg[0]; W[1] = d_fixed_msg[1]; W[2] = d_fixed_msg[2]; W[3] = d_fixed_msg[3];
    W[4] = d_fixed_msg[4]; W[5] = d_fixed_msg[5]; W[6] = d_fixed_msg[6]; W[7] = d_fixed_msg[7];
    W[8] = d_fixed_msg[8]; W[9] = d_fixed_msg[9]; W[10] = d_fixed_msg[10];
    // 中文标注: 载入变量部分和填充部分
    W[11] = m11; W[12] = m12; W[13] = m13; W[14] = padding_m14; W[15] = padding_m15; // 中文标注: 使用重命名后的填充常数

    // --- Manual Unrolling of 64 Steps ---
    // 中文标注: 手动展开 64 步

    // Steps 0-15 (use initial W values)
    // 中文标注: 前 16 步使用初始 W 值
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[0], d_K[0]); // 中文标注: 使用设备端常数 d_K
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[1], d_K[1]);
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[2], d_K[2]);
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[3], d_K[3]);
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[4], d_K[4]);
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[5], d_K[5]);
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[6], d_K[6]);
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[7], d_K[7]);
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[8], d_K[8]);
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[9], d_K[9]);
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[10], d_K[10]);
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[11], d_K[11]);
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[12], d_K[12]);
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[13], d_K[13]);
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[14], d_K[14]);
    SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[15], d_K[15]);

    // Steps 16-63 (compute new W values and use them)
    // 中文标注: 步骤 16-63，计算新的 W 值并使用，使用循环缓冲区 W[0]-W[15]
    uint32_t w; // Temp variable for new W word

    // Step 16: i=16. W[16] = sig1(W[14]) + W[9] + sig0(W[1]) + W[0]. Store at W[0].
    w = sig1(W[14]) + W[9] + sig0(W[1]) + W[0]; W[0] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[0], d_K[16]);
    // Step 17: i=17. W[17] = sig1(W[15]) + W[10] + sig0(W[2]) + W[1]. Store at W[1].
    w = sig1(W[15]) + W[10] + sig0(W[2]) + W[1]; W[1] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[1], d_K[17]);
    // Step 18: i=18. W[18] = sig1(W[2]) + W[11] + sig0(W[3]) + W[2]. Store at W[2].
    w = sig1(W[2]) + W[11] + sig0(W[3]) + W[2]; W[2] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[2], d_K[18]);
    // Step 19: i=19. W[19] = sig1(W[3]) + W[12] + sig0(W[4]) + W[3]. Store at W[3].
    w = sig1(W[3]) + W[12] + sig0(W[4]) + W[3]; W[3] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[3], d_K[19]);
    // Step 20: i=20. W[20] = sig1(W[4]) + W[13] + sig0(W[5]) + W[4]. Store at W[4].
    w = sig1(W[4]) + W[13] + sig0(W[5]) + W[4]; W[4] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[4], d_K[20]);
    // Step 21: i=21. W[21] = sig1(W[5]) + W[14] + sig0(W[6]) + W[5]. Store at W[5].
    w = sig1(W[5]) + W[14] + sig0(W[6]) + W[5]; W[5] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[5], d_K[21]);
    // Step 22: i=22. W[22] = sig1(W[6]) + W[15] + sig0(W[7]) + W[6]. Store at W[6].
    w = sig1(W[6]) + W[15] + sig0(W[7]) + W[6]; W[6] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[6], d_K[22]);
    // Step 23: i=23. W[23] = sig1(W[7]) + W[0] + sig0(W[8]) + W[7]. Store at W[7]. (Indices wrap in W)
    w = sig1(W[7]) + W[0] + sig0(W[8]) + W[7]; W[7] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[7], d_K[23]);
    // Step 24: i=24. W[24] = sig1(W[8]) + W[1] + sig0(W[9]) + W[8]. Store at W[8].
    w = sig1(W[8]) + W[1] + sig0(W[9]) + W[8]; W[8] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[8], d_K[24]);
    // Step 25: i=25. W[25] = sig1(W[9]) + W[2] + sig0(W[10]) + W[9]. Store at W[9].
    w = sig1(W[9]) + W[2] + sig0(W[10]) + W[9]; W[9] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[9], d_K[25]);
    // Step 26: i=26. W[26] = sig1(W[10]) + W[3] + sig0(W[11]) + W[10]. Store at W[10].
    w = sig1(W[10]) + W[3] + sig0(W[11]) + W[10]; W[10] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[10], d_K[26]);
    // Step 27: i=27. W[27] = sig1(W[11]) + W[4] + sig0(W[12]) + W[11]. Store at W[11].
    w = sig1(W[11]) + W[4] + sig0(W[12]) + W[11]; W[11] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[11], d_K[27]);
    // Step 28: i=28. W[28] = sig1(W[12]) + W[5] + sig0(W[13]) + W[12]. Store at W[12].
    w = sig1(W[12]) + W[5] + sig0(W[13]) + W[12]; W[12] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[12], d_K[28]);
    // Step 29: i=29. W[29] = sig1(W[13]) + W[6] + sig0(W[14]) + W[13]. Store at W[13].
    w = sig1(W[13]) + W[6] + sig0(W[14]) + W[13]; W[13] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[13], d_K[29]);
    // Step 30: i=30. W[30] = sig1(W[14]) + W[7] + sig0(W[15]) + W[14]. Store at W[14].
    w = sig1(W[14]) + W[7] + sig0(W[15]) + W[14]; W[14] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[14], d_K[30]);
    // Step 31: i=31. W[31] = sig1(W[15]) + W[8] + sig0(W[0]) + W[15]. Store at W[15].
    w = sig1(W[15]) + W[8] + sig0(W[0]) + W[15]; W[15] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[15], d_K[31]);

    // Steps 32-47
    w = sig1(W[0]) + W[9] + sig0(W[1]) + W[0]; W[0] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[0], d_K[32]);
    w = sig1(W[1]) + W[10] + sig0(W[2]) + W[1]; W[1] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[1], d_K[33]);
    w = sig1(W[2]) + W[11] + sig0(W[3]) + W[2]; W[2] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[2], d_K[34]);
    w = sig1(W[3]) + W[12] + sig0(W[4]) + W[3]; W[3] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[3], d_K[35]);
    w = sig1(W[4]) + W[13] + sig0(W[5]) + W[4]; W[4] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[4], d_K[36]);
    w = sig1(W[5]) + W[14] + sig0(W[6]) + W[5]; W[5] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[5], d_K[37]);
    w = sig1(W[6]) + W[15] + sig0(W[7]) + W[6]; W[6] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[6], d_K[38]);
    w = sig1(W[7]) + W[0] + sig0(W[8]) + W[7]; W[7] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[7], d_K[39]);
    w = sig1(W[8]) + W[1] + sig0(W[9]) + W[8]; W[8] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[8], d_K[40]);
    w = sig1(W[9]) + W[2] + sig0(W[10]) + W[9]; W[9] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[9], d_K[41]);
    w = sig1(W[10]) + W[3] + sig0(W[11]) + W[10]; W[10] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[10], d_K[42]);
    w = sig1(W[11]) + W[4] + sig0(W[12]) + W[11]; W[11] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[11], d_K[43]);
    w = sig1(W[12]) + W[5] + sig0(W[13]) + W[12]; W[12] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[12], d_K[44]);
    w = sig1(W[13]) + W[6] + sig0(W[14]) + W[13]; W[13] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[13], d_K[45]);
    w = sig1(W[14]) + W[7] + sig0(W[15]) + W[14]; W[14] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[14], d_K[46]);
    w = sig1(W[15]) + W[8] + sig0(W[0]) + W[15]; W[15] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[15], d_K[47]);

    // Steps 48-63
    w = sig1(W[0]) + W[9] + sig0(W[1]) + W[0]; W[0] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[0], d_K[48]);
    w = sig1(W[1]) + W[10] + sig0(W[2]) + W[1]; W[1] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[1], d_K[49]);
    w = sig1(W[2]) + W[11] + sig0(W[3]) + W[2]; W[2] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[2], d_K[50]);
    w = sig1(W[3]) + W[12] + sig0(W[4]) + W[3]; W[3] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[3], d_K[51]);
    w = sig1(W[4]) + W[13] + sig0(W[5]) + W[4]; W[4] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[4], d_K[52]);
    w = sig1(W[5]) + W[14] + sig0(W[6]) + W[5]; W[5] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[5], d_K[53]);
    w = sig1(W[6]) + W[15] + sig0(W[7]) + W[6]; W[6] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[6], d_K[54]);
    w = sig1(W[7]) + W[0] + sig0(W[8]) + W[7]; W[7] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[7], d_K[55]);
    w = sig1(W[8]) + W[1] + sig0(W[9]) + W[8]; W[8] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[8], d_K[56]);
    w = sig1(W[9]) + W[2] + sig0(W[10]) + W[9]; W[9] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[9], d_K[57]);
    w = sig1(W[10]) + W[3] + sig0(W[11]) + W[10]; W[10] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[10], d_K[58]);
    w = sig1(W[11]) + W[4] + sig0(W[12]) + W[11]; W[11] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[11], d_K[59]);
    w = sig1(W[12]) + W[5] + sig0(W[13]) + W[12]; W[12] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[12], d_K[60]);
    w = sig1(W[13]) + W[6] + sig0(W[14]) + W[13]; W[13] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[13], d_K[61]);
    w = sig1(W[14]) + W[7] + sig0(W[15]) + W[14]; W[14] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[14], d_K[62]);
    w = sig1(W[15]) + W[8] + sig0(W[0]) + W[15]; W[15] = w; SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[15], d_K[63]);


    // --- End of Manual Unrolling ---


    // Add the final hash values to the initial values
    // 中文标注: 将最终哈希值加到初始哈希值上
    hash.arr[0] = H0 + a; // 中文标注: 使用重命名后的初始哈希值
    hash.arr[1] = H1 + b;
    hash.arr[2] = H2 + c;
    hash.arr[3] = H3 + d;
    hash.arr[4] = H4 + e;
    hash.arr[5] = H5 + f;
    hash.arr[6] = H6 + g;
    hash.arr[7] = H7 + h;
}

// SHA-256 compression function implementation for HOST (Standard loop version)
// 中文标注: 主机端 SHA-256 压缩函数实现 (标准循环版本)
__forceinline__ __host__ void sha256_host(
    hash_t& hash, uint32_t m11, uint32_t m12, uint32_t m13)
{
    uint32_t a = H0; // 中文标注: 使用重命名后的初始哈希值
    uint32_t b = H1;
    uint32_t c = H2;
    uint32_t d = H3;
    uint32_t e = H4;
    uint32_t f = H5;
    uint32_t g = H6;
    uint32_t h = H7;

    uint32_t W[64]; // Host version can use the full W array easily

    // Load initial 16 words (from input message)
    for (int i = 0; i < 11; ++i) {
        W[i] = h_fixed_msg[i]; // 中文标注: 使用主机端常量 h_fixed_msg
    }
    W[11] = m11;
    W[12] = m12;
    W[13] = m13;
    W[14] = padding_m14; // 中文标注: 使用重命名后的填充常数
    W[15] = padding_m15;

    // Compute remaining 48 words
    for (int i = 16; i < 64; ++i) {
        W[i] = sig1(W[i - 2]) + W[i - 7] + sig0(W[i - 15]) + W[i - 16]; // 中文标注: 使用新的位操作函数
    }


    // SHA-256 Compression loop (64 steps)
    for (int i = 0; i < 64; ++i) {
        // Perform the core SHA-256 step using the macro (it's safe inside a loop too)
        SHA256_TRANSFORM_STEP(a,b,c,d,e,f,g,h, W[i], h_K[i]); // 中文标注: 使用主机端常数 h_K 和新的变换宏
    }

    // Add the final hash values to the initial values
    // 中文标注: 将最终哈希值加到初始哈希值上
    hash.arr[0] = H0 + a; // 中文标注: 使用重命名后的初始哈希值
    hash.arr[1] = H1 + b;
    hash.arr[2] = H2 + c;
    hash.arr[3] = H3 + d;
    hash.arr[4] = H4 + e;
    hash.arr[5] = H5 + f;
    hash.arr[6] = H6 + g;
    hash.arr[7] = H7 + h;
}


// Comparison, Copy, Set Worst Value (logic is the same)
// These need to be __host__ __device__ as they are used in both kernel and main
__forceinline__ __host__ __device__ bool less_than(const hash_t& lhs, const hash_t& rhs)
{
    // No PRAGMA_UNROLL needed here, it's a small loop and compiler can handle it.
    // 中文标注: 移除了原始码中关于 DEVICE_UNROLL 宏的注释
    for (int i = 0; i < 8; ++i) {
        if (lhs.arr[i] < rhs.arr[i]) {
            return true;
        } else if (rhs.arr[i] < lhs.arr[i]) {
            return false;
        }
    }
    return false;
}

__forceinline__ __host__ __device__ void copy(hash_t& dst, const hash_t& src)
{
    // Prefer memcpy if available and efficient, otherwise manual copy
#ifdef __CUDA_ARCH__
    // On device, memcpy is generally efficient
    std::memcpy(&dst, &src, sizeof(hash_t));
#else
    // On host, memcpy is standard
    std::memcpy(&dst, &src, sizeof(hash_t));
#endif
}

__forceinline__ __host__ __device__ void set_worst_hash_value(hash_t& hash)
{
    // Prefer memset if available and efficient
#ifdef __CUDA_ARCH__
    // On device, memset might be less efficient than a loop depending on context
    // A loop might be better for registers
    // No PRAGMA_UNROLL needed here either.
    for(int i=0; i<8; ++i) hash.arr[i] = 0xffffffff; // 中文标注: 使用循环设置最差哈希值，对于设备端可能更好
#else
    // On host, memset is standard and efficient
    std::memset(&hash, 0xff, sizeof(hash_t)); // 中文标注: 主机端使用 memset
#endif
}

// Base64 encoding logic (same as original, tied to the problem)
// Marked __device__ as it's primarily used in the kernel
// 中文标注: base64 编码逻辑
__device__ constexpr int base64_max = 62; // a-z, A-Z, 0-9

// 中文标注: base64 到 ascii 转换函数，现在仅标记为 __device__
__forceinline__ __device__ uint8_t base64_to_ascii(int x)
{
    assert(0 <= x && x < base64_max); // 中文标注: 添加断言
    // Using built-in hint for better performance
    __builtin_assume(0 <= x && x < 62); // 中文标注: 添加编译器提示
    if (x < 26) return 'A' + x;
    if (x < 52) return 'a' + (x - 26);
    return '0' + (x - 52); // 中文标注: base64 映射逻辑与原始码略有不同，但效果相同
}

__device__ constexpr int max_thread_count = base64_max * base64_max * base64_max * base64_max;

/// \brief Encode a value in range [0, base64_max^4) to a u32 encoded as base64.
// 中文标注: encode 函数，现在仅标记为 __device__
__forceinline__ __device__ uint32_t encode(int val)
{
    assert(0 <= val && val < max_thread_count);
    uint32_t ret{};
    for (int i = 0; i < 4; ++i) {
        ret |= (uint32_t)base64_to_ascii(val % base64_max) << (i * 8); // Note the byte order for u32
        val /= base64_max;
    }
     return ret;
}

// 中文标注: 内核函数，名称从 hash 更改为 hash_manual_unroll，并增加了 __launch_bounds__
template <int block_size>
__global__ void __launch_bounds__(block_size) hash_manual_unroll(int iteration, nonce_t* nonces)
{
    // set m11 (iteration) and m12 (thread index)
    const uint32_t m11 = encode(iteration);
    const int idx      = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t m12 = encode(idx);

    hash_t thread_best_hash{};
    set_worst_hash_value(thread_best_hash);
    uint32_t thread_best_m13{};

    // Iterate over possible values for the first 3 bytes of m13
    // These loops are small enough that manual unrolling isn't strictly needed,
    // but can be added with PRAGMA_UNROLL if desired for tiny gains.
    // PRAGMA_UNROLL // Could add here
    for (int i = 0; i < base64_max; ++i) {
    // PRAGMA_UNROLL // Could add here
        for (int j = 0; j < base64_max; ++j) {
        // PRAGMA_UNROLL // Could add here
            for (int k = 0; k < base64_max; ++k) {
                // Construct the variable part of m13, including the required 0x80 padding bit
                const uint32_t m13 = ((uint32_t)base64_to_ascii(i) << 24) |
                                     ((uint32_t)base64_to_ascii(j) << 16) |
                                     ((uint32_t)base64_to_ascii(k) << 8) |
                                     uint32_t{0x80}; // Padding bit

                hash_t current_hash;
                // Call the manually unrolled device version
                // 中文标注: 调用手动展开的设备端哈希函数
                sha256_manual_unroll_device(current_hash, m11, m12, m13);

                if (less_than(current_hash, thread_best_hash)) {
                    copy(thread_best_hash, current_hash);
                    thread_best_m13 = m13;
                }
            }
        }
    }

    // Reduce results within the block to find the best hash and corresponding m13
    // 中文标注: 块内归约，寻找块内最优哈希值和对应的 m13
    struct reduction_type_v2 { // 中文标注: 新的归约结构体，直接携带 m13
        hash_t hash;
        uint32_t m13; // Carry the m13 value along with the hash
    };

    reduction_type_v2 val_v2;
    copy(val_v2.hash, thread_best_hash);
    val_v2.m13 = thread_best_m13; // Store the best m13 found by *this* thread

    using block_reduce_v2 = hipcub::BlockReduce<reduction_type_v2, block_size>;
    __shared__ typename block_reduce_v2::TempStorage tmp_storage_v2;
    // Use the __device__ lambda with __attribute__((device))
    // 中文标注: 使用设备端 lambda 函数进行归约，并添加属性
    const reduction_type_v2 block_best_v2 =
        block_reduce_v2(tmp_storage_v2)
            .Reduce(
                val_v2,
                [] __attribute__((device))(const reduction_type_v2& lhs, const reduction_type_v2& rhs)
                    -> reduction_type_v2 { return less_than(lhs.hash, rhs.hash) ? lhs : rhs; });


    // The first thread in the block stores the block's best result
    if (threadIdx.x == 0) {
        nonces[blockIdx.x].m11 = m11;
        nonces[blockIdx.x].m12 = m12;
        nonces[blockIdx.x].m13 = block_best_v2.m13; // 中文标注: 存储归约结果中携带的最优 m13
    }
}


// Host side helper for error checking (different macro name)
// 中文标注: 主机端错误检查辅助宏，名称从 CHECK_CUDA 改为 CUDA_CHECK_ERR
#define CUDA_CHECK_ERR(call)                                                                           \
    do {                                                                                           \
        hipError_t err = call;                                                                    \
        if (err != hipSuccess) {                                                                  \
            fprintf(stderr, "CUDA error at %s:%d \"%s\"\n", __FILE__, __LINE__, hipGetErrorString(err));   \
            std::exit(EXIT_FAILURE);                                                               \
        }                                                                                          \
    } while (0)

// Host side printing functions (slightly different implementation)
// 中文标注: 主机端打印函数，名称末尾添加了 _alt
void print_u32_as_char_alt(uint32_t x, int n = 4)
{
    uint32_t tmp = swap_endian(x); // Swap for printing characters in order
    char buffer[5];
    std::memcpy(buffer, &tmp, 4);
    buffer[4] = '\0';
    for (int j = 0; j < n; ++j) {
        fputc(buffer[j], stdout); // 中文标注: 使用 fputc 替代 printf 打印字符
    }
}

// 中文标注: 主机端打印输入函数，使用 h_fixed_msg 数组
void print_input_alt(const nonce_t& nonce)
{
    for(int i=0; i<11; ++i) print_u32_as_char_alt(h_fixed_msg[i]); // 中文标注: 使用主机端常量 h_fixed_msg 打印固定消息
    print_u32_as_char_alt(nonce.m11);
    print_u32_as_char_alt(nonce.m12);
    print_u32_as_char_alt(nonce.m13, 3); // Only print the first 3 bytes of m13
    printf("\n");
}

// 中文标注: 主机端打印哈希函数
void print_hash_alt(const hash_t& hash)
{
    for (int i = 0; i < 8; ++i) {
        printf("%08x ", hash.arr[i]); // 中文标注: 使用 %08x 格式打印
    }
    printf("\n");
}

int main(int argc, char* argv[])
{
    setbuf(stdout, nullptr); // make stream unbuffered

    int iter_offset = 0;
    if (argc > 1) {
        iter_offset = std::strtol(argv[1], nullptr, 10);
    }

    // --- Copy constants to device memory ---
    // 中文标注: 将主机端常数拷贝到设备端
    CUDA_CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(d_K), h_K, sizeof(h_K)));
    CUDA_CHECK_ERR(hipMemcpyToSymbol(HIP_SYMBOL(d_fixed_msg), h_fixed_msg, sizeof(h_fixed_msg)));
    // ---------------------------------------

    hipStream_t stream;
    CUDA_CHECK_ERR(hipStreamCreate(&stream));

    hipEvent_t start, stop;
    CUDA_CHECK_ERR(hipEventCreate(&start));
    CUDA_CHECK_ERR(hipEventCreate(&stop));

    constexpr int grid_size  = 256;
    constexpr int block_size = 256;
    // Use host version for static assert
    // 中文标注: 静态断言使用主机端常量 base64_max
    static_assert(grid_size * block_size <= 62 * 62 * 62 * 62); // Use constant from host for calculation


    nonce_t* d_nonces{};
    CUDA_CHECK_ERR(hipMalloc(&d_nonces, grid_size * sizeof(nonce_t)));

    nonce_t best_nonce{};
    hash_t overall_best_hash; // 中文标注: 变量名从 best_hash 改为 overall_best_hash
    set_worst_hash_value(overall_best_hash);

    const int num_batches_to_run = INT_MAX; // Run until interrupted // 中文标注: 变量名从 num_batches 改为 num_batches_to_run
    const int iters_per_batch = 2; // Number of iterations per timing measurement // 中文标注: 变量名从 num_iters_per_batch 改为 iters_per_batch

    for (int batch_idx = 0; batch_idx < num_batches_to_run; ++batch_idx) {

        CUDA_CHECK_ERR(hipEventRecord(start, stream));

        // Launch kernel for each iteration in the batch
        for (int j = 0; j < iters_per_batch; ++j) {
            const int current_iteration = iter_offset + iters_per_batch * batch_idx + j;
            // Call the kernel using the manual unroll version
            // 中文标注: 调用新命名的内核函数 hash_manual_unroll，并传递 stream
            hash_manual_unroll<block_size><<<grid_size, block_size, 0 /* shared memory */, stream>>>(
                current_iteration, d_nonces);
            CUDA_CHECK_ERR(hipGetLastError()); // 中文标注: 在内核启动后立即检查错误
        }

        CUDA_CHECK_ERR(hipEventRecord(stop, stream));

        // Wait for the batch to complete and measure time
        CUDA_CHECK_ERR(hipEventSynchronize(stop));
        float milliseconds{}; // 中文标注: 变量名从 milliseconds 改为 milliseconds
        CUDA_CHECK_ERR(hipEventElapsedTime(&milliseconds, start, stop));

        // Calculate hash rate
        // 中文标注: 哈希率计算使用主机端常量 base64_max
        const double hashes_calculated_per_batch = static_cast<double>(iters_per_batch) * grid_size *
                                                 block_size * 62 * 62 * 62; // Use constant from host for calculation
        const double seconds = milliseconds / 1000.0;
        printf(
            "iter [%d, %d): %fGH/s (%fms)\n",
            iter_offset + iters_per_batch * batch_idx,
            iter_offset + iters_per_batch * (batch_idx + 1),
            hashes_calculated_per_batch / seconds / 1.e9, // Convert to GH/s
            milliseconds);

        // Copy block best nonces from device to host
        std::vector<nonce_t> h_block_nonces(grid_size); // 中文标注: 变量名从 h_nonces 改为 h_block_nonces
        CUDA_CHECK_ERR(hipMemcpy(
            h_block_nonces.data(), d_nonces, grid_size * sizeof(nonce_t), hipMemcpyDeviceToHost));

        // Check block best nonces and update overall best
        for (int i = 0; i < grid_size; ++i) {
            hash_t current_block_hash{};
            // Calculate hash on host using the nonce found by the block
            // Call the host version of the hash function
            // 中文标注: 主机端验证改用 sha256_host 函数
            sha256_host(current_block_hash, h_block_nonces[i].m11, h_block_nonces[i].m12, h_block_nonces[i].m13);

            if (less_than(current_block_hash, overall_best_hash)) { // 中文标注: 变量名整体更新
                best_nonce = h_block_nonces[i];
                copy(overall_best_hash, current_block_hash); // 中文标注: 变量名整体更新
                printf("Found new best:\n");
                print_input_alt(best_nonce); // 中文标注: 调用新命名的打印函数
                print_hash_alt(overall_best_hash); // 中文标注: 调用新命名的打印函数
            }
        }
    }

    printf("final result:\n");
    print_input_alt(best_nonce); // 中文标注: 调用新命名的打印函数
    print_hash_alt(overall_best_hash); // 中文标注: 调用新命名的打印函数

    CUDA_CHECK_ERR(hipFree(d_nonces)); // 中文标注: 变量名更新
    CUDA_CHECK_ERR(hipEventDestroy(stop)); // 中文标注: 变量名更新
    CUDA_CHECK_ERR(hipEventDestroy(start)); // 中文标注: 变量名更新
    CUDA_CHECK_ERR(hipStreamDestroy(stream)); // 中文标注: 变量名更新

    return 0;
}
